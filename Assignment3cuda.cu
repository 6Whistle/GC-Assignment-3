
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
using namespace std;

__global__ void addKernel(int *dev_c, const int *dev_a, const int *dev_b){
    int a_idx = threadIdx.y * blockDim.x;
    int b_idx = threadIdx.x;
    int c_idx = a_idx + b_idx;
    int sum = 0;

    for(int i = 0; i < blockDim.x; i++, a_idx++, b_idx += blockDim.x)
        sum += dev_a[a_idx] * dev_b[b_idx]; 
    dev_c[c_idx] = sum;

    return;
}

int main(void){
    const int WIDTH = 5;
    int a[WIDTH][WIDTH], b[WIDTH][WIDTH], c[WIDTH][WIDTH];
    int *dev_a, *dev_b, *dev_c;

    srand((unsigned int)time(NULL));

    for(int i = 0; i < WIDTH; i++){
        for(int j = 0; j < WIDTH; j++){
            a[i][j] = rand() % 10;
            b[i][j] = rand() % 10;
        }
    }

    hipMalloc((void **)&dev_a, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void **)&dev_b, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void **)&dev_c, WIDTH * WIDTH * sizeof(int));

    hipMemcpy((void *)dev_a, (void *)a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_b, (void *)b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);

    dim3 DimBlock(WIDTH, WIDTH);
    addKernel<<<1, DimBlock>>> (dev_c, dev_a, dev_b);

    hipMemcpy((void *)c, (void *)dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost);

    cout << "Matrix Multiplication" << endl;
    for(int i = 0; i < WIDTH; i++){
        for(int j = 0; j < WIDTH; j++)  cout << setw(4) << a[i][j];
        (i == WIDTH / 2) ? cout << "  *" : cout << "   ";

        for(int j = 0; j < WIDTH; j++)  cout << setw(4) << b[i][j];
        (i == WIDTH / 2) ? cout << "  =" : cout << "   ";

        for(int j = 0; j < WIDTH; j++)  cout << setw(4) << c[i][j];
        cout << endl;
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}